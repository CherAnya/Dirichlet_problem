#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define E 0.001
#define N  (512*512)
#define BLOCKS 256

__global__ void Kernel(double* present, double* past, double* grid, double* f, int SIZE)
{
	double h = 1.0 / (SIZE - 1);
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= SIZE && id <= (SIZE * SIZE - SIZE - 1) && id % SIZE != 0 && (id + 1) % SIZE != 0)
		present[id] = 0.25 * (past[id - SIZE] + past[id - 1] + past[id + SIZE] + past[id + 1] - pow(h, 2) * f[id]);
	else
		present[id] = past[id];
}

__global__ void Errors(double* present, double* grid, double* error)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	error[id] = fabs(present[id] - grid[id]);
}

double fr(double a, double b) {
	return 4 + 2 * pow(a, 2) - 2 * a + 2 * pow(b, 2) - 2 * b;
}

double u(double a, double b) {
	double c = 0;
	c = (pow(a, 2) - a + 1) * (pow(b, 2) - b + 1);
	return c;
}

void gran(double* arr, double* Z, int X0, int Xn, int Y0, int Yn, int SIZE)
{
	int k = 0;
	for (int i = X0; i < Xn; ++i)
	{
		for (int j = Y0; j < Yn; ++j)
		{
			k = i - X0 + j - Y0;
			arr[i * SIZE + j] = Z[k] * Z[k] - Z[k] + 1;
		}
	}

}

bool Writetofile(char* filename, double* X, double* Y, double* Z, int SIZE)
{

	FILE* fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("Can not open file %s\n", filename);
		return false;
	}
	for (int i = 0; i < SIZE; i++)
		for (int j = 0; j < SIZE; j++)
			fprintf(fp, "%7.8f %7.8f %7.8f\n", X[i], Y[j], Z[i * SIZE + j]);
	fclose(fp);
	return true;
}

int main()
{
	int SIZE = 512;
	int steperror = 1000;
	double* present = (double*)malloc((SIZE * SIZE) * sizeof(double));
	double* grid = (double*)malloc((SIZE * SIZE) * sizeof(double));
	double* past = (double*)malloc((SIZE * SIZE) * sizeof(double));


	double* X = (double*)malloc(SIZE * sizeof(double));
	double* Y = (double*)malloc(SIZE * sizeof(double));
	double* f = (double*)malloc(SIZE * SIZE * sizeof(double));
	double* errors = (double*)malloc(SIZE * SIZE * sizeof(double));

	for (int i = 0; i < SIZE; i++) {
		X[i] = double(i) / (SIZE - 1);
		Y[i] = double(i) / (SIZE - 1);
	}

	for (int i = 0; i < SIZE; i++) {
		for (int j = 0; j < SIZE; ++j) {
			past[i * SIZE + j] = 0;
			grid[i * SIZE + j] = u(X[i], Y[j]);
			f[i * SIZE + j] = fr(X[i], Y[j]);
		}
	}

	clock_t start, end;
	double time;

	start = clock();

	gran(past, X, 0, SIZE, 0, 1, SIZE);
	gran(past, X, 0, SIZE, SIZE - 1, SIZE, SIZE);
	gran(past, Y, 0, 1, 0, SIZE, SIZE);
	gran(past, Y, SIZE - 1, SIZE, 0, SIZE, SIZE);

	double localMax = 1;
	int iter = 0;

	double* dev_arrpresent = 0;
	double* dev_arrpast = 0;
	double* dev_arrgrid = 0;
	double* dev_f = 0;
	double* dev_error = 0;

	hipSetDevice(0);
	hipMalloc((void**)&dev_arrpresent, N * sizeof(double));
	hipMalloc((void**)&dev_arrpast, N * sizeof(double));
	hipMalloc((void**)&dev_arrgrid, N * sizeof(double));
	hipMalloc((void**)&dev_f, N * sizeof(double));
	hipMalloc((void**)&dev_error, N * sizeof(double));

	hipMemcpy(dev_arrpast, past, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_arrgrid, grid, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_f, f, N * sizeof(double), hipMemcpyHostToDevice);

	while (localMax > E)
	{
		Kernel << < BLOCKS, N / BLOCKS >> > (dev_arrpresent, dev_arrpast, dev_arrgrid, dev_f, SIZE);
		if (iter % steperror == 0) {
			localMax = 0;
			Errors << < BLOCKS, N / BLOCKS >> > (dev_arrpresent, dev_arrgrid, dev_error);
			hipMemcpy(errors, dev_error, N * sizeof(double), hipMemcpyDeviceToHost);
			for (int i = 0; i < SIZE * SIZE; i++)
				if (localMax < errors[i])
					localMax = errors[i];
			printf("%d, %lf\n", iter, localMax);
		}
		double* temp = dev_arrpast;
		dev_arrpast = dev_arrpresent;
		dev_arrpresent = temp;
		iter++;
	}
	hipMemcpy(past, dev_arrpresent, N * sizeof(double), hipMemcpyDeviceToHost);
	Writetofile("Jakobi.dat", X, Y, past, SIZE);

	end = clock();
	time = ((double)(end - start)) / CLOCKS_PER_SEC;

	printf("Number of iterations:\n");
	printf("%d\n", iter - 1);
	printf("Time, s:\n");
	printf("%4.2lf\n", time);

	hipFree(dev_arrpresent);
	hipFree(dev_arrpast);
	hipFree(dev_arrgrid);
	hipFree(dev_f);
	hipFree(dev_error);

	free(X);
	free(Y);
	free(present);
	free(grid);
	free(past);

	return 0;
}
